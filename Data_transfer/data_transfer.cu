
#include <hip/hip_runtime.h>
#include <stdio.h>

inline hipError_t checkCuda(hipError_t result) {
    #if defined(DEBUG) || defined(_DEBUG)
        if (result != cudaSuccess) {
            fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
            assert(result == cudaSuccess);
        }
    #endif
    return result;
}
 
__global__ void kernel(float *a, int offset) {

    int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
    float x = (float) i;
    float s = sinf(x); 
    float c = cosf(x);
    a[i] += sqrtf(s*s+c*c);

}

int main(int argc, char **argv) {

    const int TPB = 256;
    const int num_streams = 4;
    const int stream_size = 1024*sizeof(float)*TPB;
    const int N = stream_size * num_streams;
    const int streamBytes = stream_size * sizeof(float);

    float *a, *d_a;
    checkCuda(hipHostAlloc(&a, N*sizeof(float), hipHostMallocDefault));
    checkCuda(hipMalloc(&d_a, N*sizeof(float)));


    hipStream_t stream[num_streams];
    for (int i = 0; i < num_streams; ++i) {
        checkCuda(hipStreamCreate(&stream[i]));
    }

    /* SEQUENTIAL */
        memset(a, 0, N*sizeof(float));
        checkCuda(hipMemcpy(d_a, a, N*sizeof(float), hipMemcpyHostToDevice));
        kernel<<<N/TPB, TPB>>>(d_a, 0);
        checkCuda(hipMemcpy(a, d_a, N*sizeof(float), hipMemcpyDeviceToHost));

    /* VERSION 1 */
        memset(a, 0, N*sizeof(float));
        for (int i = 0; i < num_streams; ++i) {
            int offset = i * stream_size;
            checkCuda(hipMemcpyAsync(&d_a[offset], &a[offset], streamBytes, hipMemcpyHostToDevice, stream[i]) );
            kernel<<<stream_size/TPB, TPB, 0, stream[i]>>>(d_a, offset);
            checkCuda(hipMemcpyAsync(&a[offset], &d_a[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]) );
        }

    /* VERSION 2 */
        memset(a, 0, N*sizeof(float));
        for (int i = 0; i < num_streams; ++i)
        {
            int offset = i * stream_size;
            checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset], streamBytes, hipMemcpyHostToDevice, stream[i]) );
        }
        for (int i = 0; i < num_streams; ++i)
        {
            int offset = i * stream_size;
            kernel<<<stream_size/TPB, TPB, 0, stream[i]>>>(d_a, offset);
        }
        for (int i = 0; i < num_streams; ++i)
        {
            int offset = i * stream_size;
            checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]) );
        }

    for (int i = 0; i < num_streams; ++i) {
        checkCuda( hipStreamDestroy(stream[i]) );
    }
    hipFree(d_a);
    hipHostFree(a);

    return 0;
}